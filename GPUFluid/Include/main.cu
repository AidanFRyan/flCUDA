#include "SLA.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <string>
using namespace std;
using namespace SLA;
int main(){
    CuVec<double> v, *d_v;
    CuMat<double> m, *d_m;
    double* out, *d_out;

    out = new double[6];
	d_out = 0;
	d_v = 0;
	d_m = 0;

    hipMalloc((void**)d_out, sizeof(double)*6);
    hipMalloc((void**)d_m, sizeof(CuMat<double>));
    hipMalloc((void**)d_v, sizeof(CuVec<double>));

    v.reserve(4);
    m.resize(4, 6);

    v.set(1, 2);
    m.set(0, 1, 3);
	m.set(1, 3, 0.666);

    v.upload();
    m.upload();

	v.print();
	m.print();

    hipMemcpy(d_m, &m, sizeof(CuMat<double>), hipMemcpyHostToDevice);
    hipMemcpy(d_v, &v, sizeof(CuVec<double>), hipMemcpyHostToDevice);

    matvec<<<1, 1>>>(d_m, d_v, d_out);
    hipDeviceSynchronize();

    hipMemcpy(out, d_out, sizeof(double)*6, hipMemcpyDeviceToHost);
    
    for(int i = 0; i < 6; ++i){
        printf("%f\n", out[i]);
    }

    delete[] out;
    hipFree(d_out);
    m.free();
    hipFree(d_m);
    v.free();
    hipFree(d_v);

	string s;
	cin>>s;

    return 0;
}